#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<>

#define MAX_MASK_WIDTH 5
__constant__ float M[MAX_MASK_WIDTH];

__global__ void convolution1D(float* N, float* P, int MaskWidth, int Width) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	float Pvalue = 0;
	int N_start_point = i - (MaskWidth / 2);
	for (int j = 0; j < MaskWidth; j++) {
		if (N_start_point + j > 0 && N_start_point + j < Width)
			Pvalue += N[N_start_point + j] * M[j];
	}
	P[i] = Pvalue;
}

int main() {
	float* N, * P, * h_m;
	int MaskWidth, Width;
	float* d_N, * d_P;
	
	printf("Enter size of vector: ");
	scanf("%d", &Width);
	
	N = (float*)malloc(Width * sizeof(float));
	P = (float*)malloc(Width * sizeof(float));

	printf("Enter vector: ");
	for (int i = 0; i < Width; i++)
		scanf("%f", &N[i]);

	printf("Enter size of mask: ");
	scanf("%d", &MaskWidth);
	h_m = (float*)malloc(MaskWidth * sizeof(float));
	printf("Enter mask: ");
	for (int i = 0; i < MaskWidth; i++)
		scanf("%f", &h_m[i]);

	hipMalloc((void**)&d_N, Width * sizeof(float));
	hipMalloc((void**)&d_P, Width * sizeof(float));

	hipMemcpyToSymbol(HIP_SYMBOL(M), h_m, MaskWidth * sizeof(float));
	hipMemcpy(d_N, N, Width * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_P, P, Width * sizeof(float), hipMemcpyHostToDevice);

	int blocksize = 32;
	dim3 gridSize((Width + blocksize - 1) / blocksize, 1, 1);
	dim3 blockSize(blocksize, 1, 1);

	convolution1D << <gridSize, blockSize >> > (d_N, d_P, MaskWidth, Width);

	hipMemcpy(P, d_P, Width * sizeof(float), hipMemcpyDeviceToHost);

	printf("convolved vector: ");
	for (int i = 0; i < Width; i++)
		printf("%f\t", P[i]);

	hipFree(d_N);
	hipFree(d_P);
	return 0;
}