#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <>

__device__ void a(int i) {
    printf("hello");
    i = 8;
}

__global__ void kernel(int* i) {
    printf("in kernel");
    *i = 3;
    a(*i);
}

int main() {
    int* d_i;  // Pointer for GPU memory

    // Allocate memory on GPU
    hipMalloc((void**)&d_i, sizeof(int));

    // Initialize host variable
    int h_i = 5;

    // Copy host variable to GPU
    hipMemcpy(d_i, &h_i, sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel
    kernel << <1, 1 >> > (d_i);

    // Copy the result back to host
    hipMemcpy(&h_i, d_i, sizeof(int), hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_i);

    return 0;
}
