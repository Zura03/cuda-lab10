#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<>

__global__ void oddEvenSort(int* array, int n) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	for (int phase = 0; phase < n; phase++) {
		if (phase % 2 == 0) {
			if ((tid % 2 == 0) && (tid < n - 1)) {
				if (array[tid] > array[tid + 1]) {
					int temp = array[tid];
					array[tid] = array[tid + 1];
					array[tid + 1] = temp;
				}
			}
		}
		else {
			if ((tid % 2 != 0) && (tid < n - 1)) {
				if (array[tid] > array[tid + 1]) {
					int temp = array[tid];
					array[tid] = array[tid + 1];
					array[tid + 1] = temp;
				}
			}
		}
		__syncthreads();
	}
}

int main() {
	int* array, n;
	int* d_arr;

	printf("Enter n: ");
	scanf("%d", &n);

	int size = n * sizeof(int);
	array = (int*)malloc(size);

	printf("Enter the array: ");
	for (int i = 0; i < n; i++)
		scanf("%d", &array[i]);

	hipMalloc((void**)&d_arr, size);
	hipMemcpy(d_arr, array, size, hipMemcpyHostToDevice);

	dim3 gridSize(1, 1);
	dim3 blockSize(n, 1);

	oddEvenSort << <gridSize, blockSize >> > (d_arr, n);

	hipMemcpy(array, d_arr, size, hipMemcpyDeviceToHost);

	printf("Sorted array: ");
	for (int i = 0; i < n; i++)
		printf("%d\t", array[i]);

	hipFree(d_arr);
	return 0;
}