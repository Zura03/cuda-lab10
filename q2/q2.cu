/*
#include<stdio.h>
#include<stdlib.h>
#include<cuda_runtime.h>
#include<device_launch_parameters.h>

const int N = 8;

__global__ void merge(int* arr, int* temp, int left, int middle, int right) {
	int i = left;
	int j = middle;
	int k = right;

	while (i < middle && j < right) {
		if (arr[i] <= arr[j]) {
			temp[k++] = arr[i++];
		}
		else {
			temp[k++] = arr[j++];
		}
	}

	while (j < middle) {
		temp[k++] = arr[i++];
	}

	while (j < right) {
		temp[k++] = arr[j++];
	}
}
__global__ void mergesort(int* arr, int* temp, int left, int right) {
	if (right - left <= 1)
		return;

	int middle = (left + right) / 2;

	mergesort << <1, 1 >> > (arr, temp, left, middle);
	mergesort << <1, 1 >> > (arr, temp, middle, right);

	cudaDeviceSynchronize();

	merge << <1, 1 >> > (arr, temp, left, middle, right);

	cudaDeviceSynchronize();
}
int main() {
	int host_arr[N] = { 10, 27, 42,3, 9, 82, 11, 65 };

	int* d_arr, * d_temp;

	cudaMalloc((void**)&d_arr, N * sizeof(int));
	cudaMalloc((void**)d_temp, N * sizeof(int));

	cudaMemcpy(d_arr, host_arr, N * sizeof(int), cudaMemcpyHostToDevice);
	//cudaMemcpy(d_temp, host_arr, N * sizeof(int), cudaMemcpyHostToDevice);

	mergesort << <1, 1 >> > (d_arr, d_temp, 0, N);

	cudaMemcpy(host_arr, d_arr, N * sizeof(int), cudaMemcpyDeviceToHost);

	printf("sorted array: ");
	for (int i = 0; i < N; i++)
		printf("%d\t", host_arr[i]);

	cudaFree(d_arr);
	cudaFree(d_temp);
	return 0;
}*/

#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"


__device__ void merge_sequential(int* A, int m, int* B, int n, int* C) {
    int i=0, j=0, k=0;

    while(i<m && j<n) 
        if(A[i] <= B[j]) 
            C[k++] = A[i++];
        else 
            C[k++] = B[j++];
    
    if(i == m) 
        while(j < n) 
            C[k++] = B[j++];
    else 
        while(i < m) 
            C[k++] = A[i++];
}

__device__ int co_rank(int k, int* A, int m, int* B, int n) {
    int i = k < m ? k : m;
    int j = k - i;
    int i_low = 0 > (k-n) ? 0 : k-n;
    int j_low = 0 > (k-m) ? 0 : k-m;
    int delta;
    int active = 1;

    while(active) {
        if(i > 0 && j < n && A[i-1] > B[j]) {
            delta = ((i - i_low + 1) >> 1);
            j_low = j;
            j = j + delta;
            i = i - delta;
        }
        else if(j > 0 && i < m && B[j-1] >= A[i]) {
            delta = ((j - j_low + 1) >> 1);
            i_low = i;
            i = i + delta;
            j = j - delta;
        }
        else {
            active = 0;
        }
    }

    return i;
}

__global__ void merge_kernel(int* A, int m, int* B, int n, int* C) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int elementsPerThread = ceil((double)(m+n)/(blockDim.x*gridDim.x));
    int k_curr = tid * elementsPerThread;
    int k_next = min((tid + 1) * elementsPerThread, m+n);
    int i_curr = co_rank(k_curr, A, m, B, n);
    int i_next = co_rank(k_next, A, m, B, n);
    int j_curr = k_curr - i_curr;
    int j_next = k_next - i_next;

    merge_sequential(&A[i_curr], i_next-i_curr, &B[j_curr], j_next-j_curr, &C[k_curr]);
}

int main() {
    int *A, *B, *C;
    int m, n, sizeA, sizeB, sizeC;
    int *d_A, *d_B, *d_C;

    printf("Enter the size of the first array, A: ");
    scanf("%d", &m);

    printf("Enter the size of the second array, B: ");
    scanf("%d", &n);

    sizeA = sizeof(int) * m;
    sizeB = sizeof(int) * n;
    sizeC = sizeof(int) * (m+n);

    A = (int*) malloc(sizeA);
    B = (int*) malloc(sizeB);
    C = (int*) malloc(sizeC);

    printf("Enter the sorted array, A: ");
    for(int i=0; i<m; i++) 
        scanf("%d", A+i);

    printf("Enter the sorted array, B: ");
    for(int i=0; i<n; i++) 
        scanf("%d", B+i);

    hipMalloc((void**) &d_A, sizeA);
    hipMalloc((void**) &d_B, sizeB);
    hipMalloc((void**) &d_C, sizeC);
    hipMemcpy(d_A, A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, sizeB, hipMemcpyHostToDevice);

    merge_kernel<<<1, ceil((m+n)/5.0)>>>(d_A, m, d_B, n, d_C);

    hipMemcpy(C, d_C, sizeC, hipMemcpyDeviceToHost);

    printf("Resultant Array after Parallel Merge Sorting:\n");
    for(int i=0; i<m+n; i++) 
        printf("%4d", C[i]);
    printf("\n");

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    free(A);
    free(B);
    free(C);
    
    return 0;
}